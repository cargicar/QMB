#include "hip/hip_runtime.h"

/* To compile: nvcc -o main main.cu -O3 -std=c++17 -lcublas -lcusolver -Xcompiler -fopenmp
*/
#include "hip/hip_runtime.h"
//#include "device_launch_paraMeters.h"

#include<iostream>
#include <fstream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include <omp.h>

#include "hip/hip_runtime.h"
//#include "device_launch_paraMeters.h"
#include "include/cx.h"
#include <thrust/device_free.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>


#define prec_save 10

/******************************************/
/* SET Hankel MATRIX */
/******************************************/

const double g = 1;
const double w = -5;

__host__ double hxs(int s, double E, double x2) {
    if (s % 2 == 1 || s < 0) {
        return 0;
    } else if (s == 0) {
        return 1;
    } else if (s == 2) {
        return x2;
    } else {
        return (4 * (s - 3) * E * hxs(s - 4, E, x2) + (s - 3) * (s - 4) * (s - 5) * hxs(s - 6, E, x2) - 4 * w * (s - 2) * hxs(s - 2, E, x2)) / (4 * g * (s - 1));
    }
}

__device__ void dxs(double * __restrict d_x, int s, double E, double x2) {
    d_x[0]=1;
    d_x[2]=x2;
    if (s % 2 == 1 || s < 0) {
         d_x[s]=0;
    }
    else if (s > 2) {
        d_x[s]= (4 * (s - 3) * E * d_x[s - 4] + (s - 3) * (s - 4) * (s - 5) * d_x[s - 6] - 4 * w * (s - 2) * d_x[s - 2]) / (4 * g * (s - 1));
    }
}


__host__ void setHankel(double * __restrict h_h, double e, double x2, const int N) {

  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      h_h[i * N + j] = hxs(i + j, e, x2);
        }
    }
}

__global__ void gpusetHankel(double * __restrict d_h, double * __restrict d_x, double e, double x2, const int N) {

  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
     int  s = i+j;
	dxs(d_x, s, e, x2);
	d_h[i * N + j] =d_x[s];
        }
    }
}

/************************************/
/* SAVE REAL ARRAY FROM CPU TO FILE */
/************************************/
template <class T>
void saveCPUrealtxt(const T * h_in, const char *filename, const int M) {

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}

/************************************/
/* SAVE REAL ARRAY FROM GPU TO FILE */
/************************************/
template <class T>
void saveGPUrealtxt(const T * d_in, const char *filename, const int M) {

    T *h_in = (T *)malloc(M * sizeof(T));

    //    gpuErrchk(hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost));
     hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost);

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}


/************************************/
/* SAVE REAL ARRAY FROM GPU TO FILE */
/************************************/
template <class T>
bool checknan(const T * d_in, const int M) {

    T *h_in = (T *)malloc(M * sizeof(T));
    bool flag = false;
     hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost);

     for (int i = 0; i < M; i++){
       flag = isnan(h_in[i]);
       if (flag){
	 return flag;
	 break;
       }
  }
     return flag;
}

/********/
/* MAIN */
/********/
int main(){
  int N, xsize;
  double ess, xss, xlower, xupper, elower, eupper;

  std::cout << "Enter the size of the Hankel matrix: ";
  std::cin >> N;
 
  std::cout << "Enter the size of the x-grid: ";
  std::cin >> xsize;
  
  std::cout << "Enter the lower limit for x-region: ";
  std::cin >> xlower;
  std::cout << "Enter the upper limit for x-region: ";
  std::cin >> xupper;
  std::cout << "Enter the lower limit for e-region: ";
  std::cin >> elower;
  std::cout << "Enter the upper limit for e-region: ";
  std::cin >> eupper;

  const double& xinter = xupper - xlower;
  const double& einter = eupper - elower;
  // const int xs2 = 5000000; /// max xsize, fixed before compiling to avoid dynamic alloc.
  double *ees = (double *)malloc(xsize * xsize * sizeof(double));
  double *xxs = (double *)malloc(xsize * xsize * sizeof(double));
  
      /***********************/
      /* SETTING THE PROBLEM */
      /***********************/  
  // --- CUDA solver initialization
  hipsolverHandle_t solver_handle;
  hipsolverDnCreate(&solver_handle);
  // --- CUBLAS initialization
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
    
   
      #pragma omp parallel for private(xss, ess) shared(ees, xxs, N, xlower, xinter, elower, einter, xsize, g, w)
  for (int i = 0; i < xsize; ++i) {
    xss = xlower + i * xinter / xsize;
    for (int j = 0; j < xsize; ++j) {
      ess = elower + j * einter / xsize;
   
      // --- Allocate device space for the input matrix 
     
      // thrust::device_vector<double> dh(N * N * sizeof(double) ); // GPU hankel buffer 
      // double *dhptr = thrust::raw_pointer_cast(&dh[0]); // get hankel pointer
    thrust::device_ptr<double> device_h_ptr = thrust::device_malloc<double>(N * N * sizeof(double));
    double * dhptr = thrust::raw_pointer_cast(device_h_ptr);
       // --- Allocate device space for xs 
      //  double *d_h; hipMalloc(&d_h, N * N * sizeof(double));
      thrust::device_vector<double> dx(N * sizeof(double) ); // GPU xs buffer 
      double *dxptr = thrust::raw_pointer_cast(&dx[0]); // get dx pointer
   
      
         
      /****************************************/
      /* COMPUTING THE CHOLESKY DECOMPOSITION */
      /****************************************/
      // --- cuSOLVE input/output parameters/arrays
      int work_size = 0;
      int *devInfo;     hipMalloc(&devInfo, sizeof(int));

      int threads = 256;
      int blocks = (N+threads-1)/threads;  // ensure threads*blocks ≥ steps

      gpusetHankel<<<blocks,threads>>>(dhptr, dxptr,  ess, xss, N);

      // --- CUDA CHOLESKY initialization
      hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, dhptr, N, &work_size);
    // --- CUDA POTRF execution
      double *work;   hipMalloc(&work, work_size * sizeof(double));
      hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, dhptr, N, work, work_size, devInfo);    
      // int devInfo_h = 0;  hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
     bool flag = checknan(dhptr, N * N);
     //	 std::cout << "flag :" << flag << "\n";
        if (!flag){
	  ees[i*xsize+j]=ess;
	  xxs[i*xsize+j]=xss;  
      }
      // --- At this point, the lower triangular part of A contains the elements of L. 
      /***************************************/
      /* CHECKING THE CHOLESKY DECOMPOSITION */
      /***************************************/
      
      //saveCPUrealtxt(h_h, "solveSquareLinearSystemCholeskyCUDA\\h_A.txt", N * N);
      //saveGPUrealtxt(d_h, "solveSquareLinearSystemCholeskyCUDA\\d_A.txt", N * N);
      
	 // free memory
       	//hipFree(dh);
    thrust::device_free(device_h_ptr);
    }
  }
   hipsolverDnDestroy(solver_handle);              
    std::ofstream xsfile("xs.txt");
    std::ofstream esfile("es.txt");
    std::ofstream gridfile("grid.txt");
   
    gridfile << N << " " << xsize << std::endl;
   
    for (int i=0; i < xsize; ++i){
        for (int j = 0; j < xsize; ++j) {
	  xsfile<< xxs[i*xsize+j] <<" "<< std::endl;
	  esfile<< ees[i*xsize+j] <<" "<< std::endl;
	}
    }
    xsfile.close();
    esfile.close();
    gridfile.close();
    delete[] ees;
    delete[] xxs; 

//  cusolveSafeCall(hipsolverDnDestroy(solver_handle));
    return 0;

}
